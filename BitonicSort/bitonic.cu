#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "wb.h"
#include <iostream>
using namespace std;

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 512 // 2^9
#define BLOCKS 128 // 2^15
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj;
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  if ((ixj)>i) {
    if ((i&k)==0) {
      if (dev_values[i]>dev_values[ixj]) {
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      if (dev_values[i]<dev_values[ixj]) {
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);    
  dim3 threads(THREADS,1); 

  int j, k;

  for (k = 2; k <= NUM_VALS; k <<= 1) {
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
}

int main(int argc, char **argv)
{
	wbArg_t args;
    args = wbArg_read(argc, argv);
  	clock_t start, stop;

  	int inputLength;
    float  *data;
    data = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	
	cout<<"No: of Values : "<<inputLength<<endl;
	
	start = clock();
	bitonic_sort(data); 
	stop = clock();

	wbSolution(args, data, inputLength);
	print_elapsed(start, stop);
}

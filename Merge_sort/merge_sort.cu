#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <sys/time.h>
#include "wb.h"
using namespace std;

long readList(long**);

void mergesort(float*, dim3, dim3);
__global__ void gpu_mergesort(float*, float*, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(float*, float*, long, long, long);

#define min(a, b) (a < b ? a : b)
#define size 10000

bool verbose;
int main(int argc, char** argv) {
    wbArg_t args;
    args = wbArg_read(argc, argv);
    clock_t start,end;
    double cput;

    start = clock();

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    /*for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-' && argv[i][1] && !argv[i][2]) {
            char arg = argv[i][1];
            unsigned int* toSet = 0;
            switch(arg) {
                case 'x':
                    toSet = &threadsPerBlock.x;
                    break;
                case 'y':
                    toSet = &threadsPerBlock.y;
                    break;
                case 'z':
                    toSet = &threadsPerBlock.z;
                    break;
                case 'X':
                    toSet = &blocksPerGrid.x;
                    break;
                case 'Y':
                    toSet = &blocksPerGrid.y;
                    break;
                case 'Z':
                    toSet = &blocksPerGrid.z;
                    break;
                case 'v':
                    verbose = true;
                    break;
                default:
                    cout << "unknown argument: " << arg << '\n';
                    return -1;
            }

            if (toSet) {
                i++;
                *toSet = (unsigned int) strtol(argv[i], 0, 10);
            }
        }
        else {
            if (argv[i][0] == '?' && !argv[i][1])
                cout << "help:\n";
            else
                cout << "invalid argument: " << argv[i] << '\n';
            return -1;
        }
    }
    */

    if (verbose) {
        cout << "\nthreadsPerBlock:"
                  << "\n  x: " << threadsPerBlock.x
                  << "\n  y: " << threadsPerBlock.y
                  << "\n  z: " << threadsPerBlock.z
                  << "\n\nblocksPerGrid:"
                  << "\n  x:" << blocksPerGrid.x
                  << "\n  y:" << blocksPerGrid.y
                  << "\n  z:" << blocksPerGrid.z
                  << "\n\n total threads: " 
                  << threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                     blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z
                  << "\n\n";
                  
               
    }
int inputLength;
 float  *data;
data = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
   
   
    
    

    mergesort(data, threadsPerBlock, blocksPerGrid);
    
    
   
    wbSolution(args, data, inputLength);

    cout<<"Sorting "<<size<<" Numbers : ";
    /*for (int i = 0; i < size; i++) {
        cout << data[i] << " ";
    } 
    cout<<"\n";
*/
    end = clock();
    cput = ((double)(end-start))/CLOCKS_PER_SEC;
    cout<<"Running time = " << cput << endl;
}

void mergesort(float * data, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    float* D_data;
    float* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    checkCudaErrors(hipMalloc((void**) &D_data, size * sizeof(long)));
    checkCudaErrors(hipMalloc((void**) &D_swp, size * sizeof(long)));

    checkCudaErrors(hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice));
 
    checkCudaErrors(hipMalloc((void**) &D_threads, sizeof(dim3)));
    checkCudaErrors(hipMalloc((void**) &D_blocks, sizeof(dim3)));

    checkCudaErrors(hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

    float* A = D_data;
    float* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        if (verbose) {
            cout << "mergeSort - width: " << width 
                      << ", slices: " << slices 
                      << ", nThreads: " << nThreads << '\n';
        }

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    checkCudaErrors(hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost));
    
        checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
}

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}


__global__ void gpu_mergesort(float* source, float* dest, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(float* source, float* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}
